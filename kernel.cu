
#include <hip/hip_runtime.h>
#include <iostream>
const int PALETTE_SIZE = 16;

//
__constant__ //bloque de memoria en la GPU
unsigned int d_Pallete[PALETTE_SIZE];

__device__
unsigned int divergente(double cx, double cy, int max_iterations) {

    int iter = 0;

    double vx = cx;
    double vy = cy;

    while(iter<max_iterations && (vx*vx+vy*vy)<=4) {
        //Zn+1 = Zn^2 + C
        double tx = vx * vx - vy * vy + cx; //vx^2-vy^2+cx
        double ty = 2 * vx * vy + cy; // 2 vx vy + cy

        vx = tx;
        vy = ty;

        iter++;
    }

    if(iter>0 && iter<max_iterations) {
        // diverge
        int color_idx = iter % PALETTE_SIZE;
        return d_Pallete[color_idx];
    }

    if((vx*vx+vy*vy)>4) {
        return d_Pallete[0];
    }

    // convergente
    return 0;
}

extern "C"
void copy_pallete(unsigned int* h_pallete)
{
    hipMemcpyToSymbol(HIP_SYMBOL(d_Pallete), h_pallete, PALETTE_SIZE*sizeof(unsigned int));
}

__global__
void mandelbrotKernel(unsigned int* buffer,
unsigned int width, unsigned height
, double x_min, double x_max, double y_min,
double y_max, double dx, double dy, int max_iterations)
{
    // double dx = (x_max-x_min) / WIDTH;
    // double dy = (y_max-y_min) / HEIGHT;

    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < width*height)
    {
        int j = id/width;
        int i = id%width;

        double x = x_min + i*dx;
        double y = y_max - j*dy;

        // C = X+Yi
        unsigned int color = divergente(x,y, max_iterations);
        buffer[j*width + i] = color;
    }
}

extern "C"
void mandelbrotGPUkernel(unsigned int* buffer,
unsigned int width, unsigned height
, double x_min, double x_max, double y_min,
double y_max, int max_iterations)
{
    double dx = (x_max-x_min) / width;
    double dy = (y_max-y_min) / height;

    int threds_per_block = 1024;
    int blocks_in_grid = std::ceil(float(width*height/threds_per_block));

    mandelbrotKernel<<<blocks_in_grid, threds_per_block>>>(buffer,
        width, height,
        x_min, x_max, y_min, y_max,
        dx, dy,
        max_iterations);


}